#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/device_functions.h>
#include ""
#include<math.h>
#include <stdio.h>
#include<time.h>
#include <iostream>
#include <stdlib.h>
#include "GpuTimer.h"
using namespace std;

#define BLOCK_SIZE 16
#define TILE_WIDTH BLOCK_SIZE    //since the tile is of BLOCK_SIZE elements in each direction

//Compute C=A*B
// Serial implementation for running on CPU using a single thread.

void MatrixMultiplyCpu(float* A, float* B, float* C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns)
{
	for (int i = 0; i < numARows; i++) {
		for (int j = 0; j < numBColumns; j++) {
			float Cvalue = 0;
			for (int k = 0; k < numAColumns; k++) {
				Cvalue += A[i*numAColumns + k] * B[k*numBColumns + j];
			}
			C[i*numCColumns + j] = Cvalue;
		}
	}
}


//GPU Kernel for Tiled Matrix Multiplication

__global__ void TiledMatrixMultiply(float* A, float* B, float* C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns)
{
	__shared__ float ds_A[TILE_WIDTH][TILE_WIDTH];       
	__shared__ float ds_B[TILE_WIDTH][TILE_WIDTH];

	int Row = blockIdx.y * blockDim.y + threadIdx.y;      //calculate row index
	int Col = blockIdx.x * blockDim.x + threadIdx.x;      //calculate column index
	
	int n = numAColumns - 1;
	
	float Cvalue = 0;

	for (int p = 0; p < n / TILE_WIDTH + 1; ++p) {               // where p is the phase

		if (p * TILE_WIDTH + threadIdx.x < numAColumns && Row < numARows)
			ds_A[threadIdx.y][threadIdx.x] = A[Row*numAColumns + p*TILE_WIDTH + threadIdx.x];
		else
			ds_A[threadIdx.y][threadIdx.x] = 0.0;

		if (p * TILE_WIDTH + threadIdx.y < numBColumns && Col < numBColumns)
			ds_B[threadIdx.y][threadIdx.x] = B[(p*TILE_WIDTH + threadIdx.y)*numBColumns + Col];
		else
			ds_B[threadIdx.y][threadIdx.x] = 0.0;

		__syncthreads();

		if (Row < numARows && Col < numBColumns)
			for (int k = 0; k < TILE_WIDTH; ++k) {
				Cvalue += ds_A[threadIdx.y][k] * ds_B[k][threadIdx.x];
				__syncthreads();
			}
	}

	if (Row < numCRows && Col < numCColumns)
		C[Row*numCColumns + Col] = Cvalue;

}

int main(void)
{
	hipError_t err = hipSuccess;

	int numARows = 960; // number of rows in the matrix A
	int numAColumns = 640; // number of columns in the matrix A
	int numBRows = 640; // number of rows in the matrix B
	int numBColumns = 800; // number of columns in the matrix B

	int numCRows; // number of rows in the matrix C (you have to set this)
	int numCColumns; // number of columns in the matrix C (you have to set this)

					 //@@ Insert Your Code Here to Set numCRows and numCColumns

	numCRows = numARows;
	numCColumns = numBColumns;

	//Allocate the host memory for the input and output matrices

	float *h_A = (float *)malloc(sizeof(float)*numARows*numAColumns);
	float *h_B = (float *)malloc(sizeof(float)*numBRows*numBColumns);
	float *h_C = (float *)malloc(sizeof(float)*numCRows*numCColumns);
	float *h_C_CPU = (float *)malloc(sizeof(float)*numCRows*numCColumns);

	//Random Initialize Matrix A. 
	//There are several ways to do this, such as making functions for manual input or using random numbers. 
	//In this case, we simply use a for loop to fill the cells with trigonometric values of the indices:
	// Set the Seed for the random number generator rand() 
	//srand(clock());

	for (int i = 0; i<numARows; i++)
	{
		for (int j = 0; j<numAColumns; j++)
		{
			//h_A[i*numAColumns+j]=(float)rand() /(float)(RAND_MAX)*4.0;
			h_A[i*numAColumns + j] = sin(i);
		}
	}

	//Initialize Matrix B

	for (int i = 0; i<numBRows; i++)
	{
		for (int j = 0; j<numBColumns; j++)
		{
			//h_B[i*numBColumns+j]=(float)rand() /(float)(RAND_MAX) *4.0;
			h_B[i*numBColumns + j] = cos(j);

		}
	}

	//Allocate memory on the device for input and output matrices and record the needed time

	float *d_A, *d_B, *d_C;
	GpuTimer timer;
	timer.Start();

	//@@Insert Your Code Here to allocate memory for d_A, d_B, d_C

	float sizeA = numARows * numAColumns * sizeof(float);
	float sizeB = numBRows * numBColumns * sizeof(float);
	float sizeC = numCRows * numCColumns * sizeof(float);

	err = hipMalloc((void **)&d_A, sizeA * sizeof(float));

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device A (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&d_B, sizeB * sizeof(float));

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device B (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&d_C, sizeC * sizeof(float));

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device C (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	timer.Stop();
	printf("Time to allocate memory on the device is: %f msecs.\n", timer.Elapsed());



	//Copy the input matrices A and B from the host to the device and record the needed time

	GpuTimer timer1;
	timer1.Start();

	//@@ Insert Your Code Here to copy matrices A and B from Host to Device

	hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice);

	timer1.Stop();
	printf("Time to copy the Matrix from the host to the device is: %f msecs.\n", timer1.Elapsed());


	//Do the Processing on the GPU
	//@@ Insert Kernel Execution Configuration Parameters

	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
	dim3 gridDim((numCColumns - 1) / BLOCK_SIZE + 1, (numCRows - 1) / BLOCK_SIZE + 1, 1);

	//Invoke the TiledMatrixMultiply kernel and record the needed time for its execution

	GpuTimer timer2;
	timer2.Start();

	//@@ Insert Your Code Here for Kernel Invocation

	TiledMatrixMultiply << < gridDim, dimBlock >> > (d_A, d_B, d_C, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);

	timer2.Stop();
	printf("Implemented CUDA code ran in: %f msecs.\n", timer2.Elapsed());

	//Copy resulting matrix from device to host and record the needed time

	GpuTimer timer3;
	timer3.Start();

	//@@ Insert Your Code Here to Copy the resulting Matrix d_C from device to the Host h_C

	hipMemcpy(h_C, d_C, sizeC, hipMemcpyDeviceToHost);

	timer3.Stop();
	printf("Time to copy the resulting Matrix from the device to the host is: %f msecs.\n", timer3.Elapsed());


	//Do the Processing on the CPU

	clock_t begin = clock();

	//@@ Insert Your Code Here to call the CPU function MatrixMultiplyCpu where the resulting matrix is h_C_CPU

	MatrixMultiplyCpu(h_A, h_B, h_C_CPU, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);

	clock_t end = clock();
	double time_spent = (double)(end - begin) / CLOCKS_PER_SEC * 1000;
	printf("Implemented CPU serial code ran in: %f msecs.\n", time_spent);

	//Verify Results Computed by GPU and CPU

	for (int i = 0; i<numCRows; i++)
	{
		for (int j = 0; j<numCColumns; j++)
		{
			if (fabs(h_C_CPU[i*numCColumns + j] - h_C[i*numCColumns + j]) > 1e-2)
			{
				fprintf(stderr, "Result verification failed at element (%d,%d)!\n", i, j);
				exit(EXIT_FAILURE);
			}
		}
	}
	printf("Test PASSED\n");


	//Free host memory
	free(h_A);
	free(h_B);
	free(h_C);
	free(h_C_CPU);

	//Free device memory
	//@@ Insert Your Code Here to Free Device Memory

	free(d_A);
	free(d_B);
	free(d_C);

	return 0;

}